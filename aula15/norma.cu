#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
//INSIRA AS IMPORTACOES NECESSARIAS DA THRUST
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
// FIM
#include <cmath>
#include <random>

using namespace std::chrono;

void reportTime(const char* msg, steady_clock::duration span) {
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " <<
    ms.count() << " milisegundos" << std::endl;
};

 // CRIE UMA FUNCTOR PARA CALCULAR A SQUARE
struct square
{
    __host__ __device__
    double operator()(const int& x) {
           return x*x;
    }
};

// IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
float magnitude(thrust::device_vector<int> v) {
    float result;

    thrust::transform(v.begin(), v.end(), v.begin(), square());
    result = thrust::reduce(v.begin(), v.end());

    return std::sqrt(result);
    // return std::sqrt(thrust::transform_reduce(v.begin(), v.end(), square(), 0.0, plus()));
};

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    steady_clock::time_point ts, te;

    // Faça um  vector em thrust 
    thrust::device_vector<int> v_d(n);

    // inicilize o  vector
    ts = steady_clock::now();

    std::generate(v_d.begin(), v_d.end(), std::rand);


    te = steady_clock::now();
    reportTime("Inicializacao", te - ts);

    // Calcule a magnitude do vetor
    ts = steady_clock::now();
    float len = magnitude(v_d);
    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);


    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;
}